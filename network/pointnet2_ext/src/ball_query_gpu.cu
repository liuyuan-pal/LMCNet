#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int nsample,
                                        const float *__restrict__ new_xyz,
                                        const float *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 3 + 0];
    float new_y = new_xyz[j * 3 + 1];
    float new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float x = xyz[k * 3 + 0];
      float y = xyz[k * 3 + 1];
      float z = xyz[k * 3 + 2];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_point_kernel_wrapper(int b, int n, int m, float radius,
                                     int nsample, const float *new_xyz,
                                     const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}

// input: new_xyz(b, m, f) xyz(b, n, f)
// output: idx(b, m, nsample)
__global__ void query_ball_feats_kernel(int b, int n, int m,  int f, float radius,
                                        int nsample,
                                        const float *__restrict__ new_xyz,
                                        const float *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * f;
  new_xyz += batch_index * m * f;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    const float* new_xyz_cur = &new_xyz[j * f];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      const float* xyz_cur = &xyz[k * f];
      float d2=0.f;
      for (int fi=0; fi<f; ++fi) d2+=(xyz_cur[fi]-new_xyz_cur[fi])*(xyz_cur[fi]-new_xyz_cur[fi]);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_feats_kernel_wrapper(int b, int n, int m, int f, float radius,
                                     int nsample, const float *new_xyz,
                                     const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_feats_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, f, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}
